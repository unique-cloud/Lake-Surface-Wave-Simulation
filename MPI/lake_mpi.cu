#include "hip/hip_runtime.h"
#include "mpi.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define _USE_MATH_DEFINES

#define XMIN 0.0
#define XMAX 1.0
#define YMIN 0.0
#define YMAX 1.0

#define MAX_PSZ 10
#define TSCALE 1.0
//#define VSQR 0.1

void init(double* u, double* pebbles, int n);
// void evolve(double* un, double* uc, double* uo, double* pebbles, int n, double h, double dt, double t);
int tpdt(double* t, double dt, double end_time);
void print_heatmap(const char* filename, double* u, int n, double h);
void print_heatmap_part(const char* filename, double* u, int n, double h, int start, int height);
void init_pebbles(double* p, int pn, int n);

// void run_cpu(double* u, double* u0, double* u1, double* pebbles, int n, double h, double end_time);

extern void run_gpu(double* u, double* u0, double* u1, double* pebbles, int n, double h, double end_time, int nthreads);
extern void run_multi_gpu(double* u, double* u0, double* u1, double* pebbles, int n, int m, double h, double end_time, int nthreads, int rank, int size);

int main(int argc, char* argv[]) {
    int rank;
    int numproc;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &numproc);

    if (argc != 5) {
        printf("Usage: %s npoints npebs time_finish nthreads \n", argv[0]);
        return -1;
    }

    if (numproc != 4) {
        printf("This program shouble be run in exactly 4 nodes, you have %d\n", numproc);
        return -1;
    }

    int npoints = atoi(argv[1]);
    int npebs = atoi(argv[2]);
    double end_time = (double)atof(argv[3]);
    int nthreads = atoi(argv[4]);
    int narea = npoints * npoints;

    double *u_i0, *u_i1;
    double /**u_cpu, */ *u_gpu, *pebs;
    double h;

    double /*elapsed_cpu, */ elapsed_gpu;
    struct timeval /*cpu_start, cpu_end, */ gpu_start, gpu_end;

    u_i0 = (double*)malloc(sizeof(double) * narea);
    u_i1 = (double*)malloc(sizeof(double) * narea);
    pebs = (double*)malloc(sizeof(double) * narea);

    // u_cpu = (double*)malloc(sizeof(double) * narea);
    u_gpu = (double*)malloc(sizeof(double) * narea);

    if (rank == 0)
        printf("Running %s with (%d x %d) grid, until %f, with %d threads\n", argv[0], npoints, npoints, end_time, nthreads);

    h = (XMAX - XMIN) / npoints;

    int return_status;
    if (rank == 0)
        init_pebbles(pebs, npebs, npoints);
    return_status = MPI_Bcast((void*)pebs, narea, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    if (return_status != MPI_SUCCESS)
        return return_status;

    init(u_i0, pebs, npoints);
    init(u_i1, pebs, npoints);

    if (rank == 0)
        print_heatmap("lake_i.dat", u_i0, npoints, h);

    /*gettimeofday(&cpu_start, NULL);
    run_cpu(u_cpu, u_i0, u_i1, pebs, npoints, h, end_time);
    gettimeofday(&cpu_end, NULL);
    elapsed_cpu = ((cpu_end.tv_sec + cpu_end.tv_usec * 1e-6) - (cpu_start.tv_sec + cpu_start.tv_usec * 1e-6));
    printf("CPU took %f seconds\n", elapsed_cpu);*/

    gettimeofday(&gpu_start, NULL);
    // calculate the height of a block
    int height = npoints / numproc;
    double* u_i0_block = u_i0 + rank * height * npoints;
    double* u_i1_block = u_i1 + rank * height * npoints;
    double* pebs_block = pebs + rank * height * npoints;
    run_multi_gpu(u_gpu, u_i0_block, u_i1_block, pebs_block, npoints, height, h, end_time, nthreads, rank, numproc);
    gettimeofday(&gpu_end, NULL);
    elapsed_gpu = ((gpu_end.tv_sec + gpu_end.tv_usec * 1e-6) - (gpu_start.tv_sec + gpu_start.tv_usec * 1e-6));
    printf("GPU took %f seconds\n", elapsed_gpu);

    char filename[15];
    sprintf(filename, "lake_f_%d.dat", rank);
    /*print_heatmap("lake_f_cpu.dat", u_cpu, npoints, h);
    print_heatmap("lake_f_gpu.dat", u_gpu, npoints, h);*/
    print_heatmap_part(filename, u_gpu, npoints, h, height * rank, height);
    free(u_i0);
    free(u_i1);
    free(pebs);
    // free(u_cpu);
    free(u_gpu);

    MPI_Finalize();

    return 0;
}

/*void run_cpu(double* u, double* u0, double* u1, double* pebbles, int n, double h, double end_time) {
    double *un, *uc, *uo;
    double t, dt;

    un = (double*)malloc(sizeof(double) * n * n);
    uc = (double*)malloc(sizeof(double) * n * n);
    uo = (double*)malloc(sizeof(double) * n * n);

    memcpy(uo, u0, sizeof(double) * n * n);
    memcpy(uc, u1, sizeof(double) * n * n);

    t = 0.;
    dt = h / 2.;

    while (1) {
        evolve(un, uc, uo, pebbles, n, h, dt, t);

        memcpy(uo, uc, sizeof(double) * n * n);
        memcpy(uc, un, sizeof(double) * n * n);

        if (!tpdt(&t, dt, end_time))
            break;
    }

    memcpy(u, un, sizeof(double) * n * n);
}*/

void init_pebbles(double* p, int pn, int n) {
    int i, j, k, idx;
    int sz;

    srand(time(NULL));
    memset(p, 0, sizeof(double) * n * n);

    for (k = 0; k < pn; k++) {
        i = rand() % (n - 4) + 2;
        j = rand() % (n - 4) + 2;
        sz = rand() % MAX_PSZ;
        idx = j + i * n;
        p[idx] = (double)sz;
    }
}

double f(double p, double t) { return -expf(-TSCALE * t) * p; }

int tpdt(double* t, double dt, double tf) {
    if ((*t) + dt > tf)
        return 0;
    (*t) = (*t) + dt;
    return 1;
}

void init(double* u, double* pebbles, int n) {
    int i, j, idx;

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            idx = j + i * n;
            u[idx] = f(pebbles[idx], 0.0);
        }
    }
}

/*void evolve(double* un, double* uc, double* uo, double* pebbles, int n, double h, double dt, double t) {
    int i, j, idx;

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            idx = j + i * n;

            if (i == 0 || i == n - 1 || j == 0 || j == n - 1) {
                un[idx] = 0.;
            } else {
                // un[idx] = 2 * uc[idx] - uo[idx] + VSQR * (dt * dt) * ((uc[idx - 1] + uc[idx + 1] + uc[idx + n] + uc[idx - n] - 4 * uc[idx]) / (h * h) + f(pebbles[idx], t));
                un[idx] = 2 * uc[idx] - uo[idx] + VSQR * (dt * dt) * ((uc[idx - 1] + uc[idx + 1] + uc[idx + n] + uc[idx - n] + 0.25 * (uc[idx - 1 - n] + uc[idx - 1 + n] + uc[idx + 1 - n] + uc[idx + 1 + n]) - 5 * uc[idx]) / (h * h) + f(pebbles[idx], t));
            }
        }
    }
}*/

void print_heatmap(const char* filename, double* u, int n, double h) {
    int i, j, idx;

    FILE* fp = fopen(filename, "w");

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            idx = j + i * n;
            fprintf(fp, "%f %f %f\n", i * h, j * h, u[idx]);
        }
    }

    fclose(fp);
}

void print_heatmap_part(const char* filename, double* u, int n, double h, int start, int height) {
    int i, j, idx;

    FILE* fp = fopen(filename, "w");

    for (i = start; i < start + height; i++) {
        for (j = 0; j < n; j++) {
            idx = j + i * n;
            fprintf(fp, "%f %f %f\n", i * h, j * h, u[idx - start * n]);
        }
    }

    fclose(fp);
}
